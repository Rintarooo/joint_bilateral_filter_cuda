#include "hip/hip_runtime.h"
#include "joint_bilateral_filter.cuh"

static texture<float4, hipTextureType2D, hipReadModeElementType> jointTex;
static texture<float, hipTextureType2D, hipReadModeElementType> srcTex;

static __global__ void applyJointBilateral(float* dst, float sigma_color, float sigma_spatial, int radius, int rows, int cols)
{
  const int u_ = blockIdx.x * blockDim.x + threadIdx.x;
  const int v_ = blockIdx.y * blockDim.y + threadIdx.y;
  const int i  = u_ + v_*cols;
  
  if(u_ < radius || cols-radius <= u_ || v_ < radius || rows-radius <= v_){
    dst[i] = 0.0;
  }
  else{
    // const float4 p = tex2D(jointTex, u_+0.5, v_+0.5);
    const float p_dash = tex2D(srcTex, u_+0.5, v_+0.5);
              
    float sum = 0.0;
    float sumw = 0.0;
    const float den_spatial = 2*sigma_spatial*sigma_spatial;
    const float den_color = 2*sigma_color*sigma_color; 
    for(int uu = -radius; uu <= radius; uu++){
        for(int vv = -radius; vv <= radius; vv++){
              const float w_spatial = __expf(-(uu*uu + vv*vv) / den_spatial);
              // const float4 q = tex2D(jointTex, u_+uu+0.5, v_+vv+0.5);
              // const float id = (fabsf(p.x-q.x) + fabsf(p.y-q.y) + fabsf(p.z-q.z))/3.0;
              // const float id = fabsf(p.x-q.x);
              const float src_depth = tex2D(srcTex, u_+uu+0.5, v_+vv+0.5);
              const float id = fabsf(p_dash-src_depth);
              const float w_color = __expf(-(id*id) / den_color);
              const float w = w_spatial * w_color;
              // const float src_depth = tex2D(srcTex, u_+uu+0.5, v_+vv+0.5);
              // const float src_depth = src[i];
              sum += w * src_depth;
              sumw += w;
        }
     }
    dst[i] = sum / sumw;
    // const float src_depth = tex2D(srcTex, u_+0.5, v_+0.5);
    // dst[i] = src_depth;
  }
  // const float4 p = tex2D(jointTex, u_+0.5, v_+0.5);
  // dst[i] = (p.x+p.y+p.z)/3.0;    
}


void applyJointBilateralCaller(float*src, float* dst, float4* joint, int rows, int cols, int imageStep_src, int imageStep_joint,
  float sigma_color, float sigma_spatial, int radius)
{
  // TODO set dimBlock based on warp size
  dim3 dimBlock(16, 16);
  dim3 dimGrid((cols  + dimBlock.x - 1) / dimBlock.x,
               (rows + dimBlock.y - 1) / dimBlock.y);
 
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); 
  jointTex.normalized     = false;
  jointTex.addressMode[0] = hipAddressModeClamp;  // out of border references return first or last element
  jointTex.addressMode[1] = hipAddressModeClamp;
  jointTex.filterMode     = hipFilterModeLinear;
  CV_CUDEV_SAFE_CALL(hipBindTexture2D(0, jointTex, joint, channelDesc, cols, rows, imageStep_joint));
  
  hipChannelFormatDesc channelDesc_float1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); 
  srcTex.normalized     = false;
  srcTex.addressMode[0] = hipAddressModeClamp;  // out of border references return first or last element
  srcTex.addressMode[1] = hipAddressModeClamp;
  srcTex.filterMode     = hipFilterModeLinear;
  CV_CUDEV_SAFE_CALL(hipBindTexture2D(0, srcTex, src, channelDesc_float1, cols, rows, imageStep_src));

  CV_CUDEV_SAFE_CALL(hipGetLastError());
  CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());

  applyJointBilateral<<<dimGrid, dimBlock>>>(dst, sigma_color, sigma_spatial, radius, rows, cols);

  CV_CUDEV_SAFE_CALL(hipGetLastError());
  CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
  CV_CUDEV_SAFE_CALL(hipUnbindTexture(jointTex));
  CV_CUDEV_SAFE_CALL(hipUnbindTexture(srcTex));

}